
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#define vector2D std::vector<std::vector<int>>
#define vector1D std::vector<int>
#define MAXNumber 10
#define MINNumber 0

__global__ void multiplication(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tmpSum = 0;
    if (row< n && col < n) {
        for (int i = 0; i < n; i++) {
            tmpSum += A[row * n + i] * B[i * n + col];
        }
    }
    C[row * n + col] = tmpSum;
}
void translate(vector2D T,vector1D &arrayT,int n){
    for(int i=0;i<n;i++)
        for(int j=0;j<n;j++)
            arrayT[i*n+j]=T[i][j];

}
void display(vector2D t){
    int n = t.size();
    int m = t[0].size();

    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++)
            std::cout<<t[i][j]<<" ";
        std::cout<<std::endl;
    }
}
void display(vector1D t){
    int n = t.size();
    for(int i=0;i<n;i++){
        std::cout<<t[i]<<" ";
    }
    std::cout<<std::endl;
}

void multiplyCUDA(vector2D &A, vector2D &B,vector2D &C, int n) {
    int size = n*n;
    vector1D arrayA(size,0);
    vector1D arrayB(size,0);
    vector1D arrayC(size,0);

    translate(A,arrayA,n); // przepisanie danych z wektora wektów to pojedyńczego wektora
    translate(B,arrayB,n);
    translate(C,arrayC,n);
    display(arrayA);
    display(arrayB);
    display(arrayC);
    
    size_t sizeT = size * sizeof(int); 
    int *d_A,*d_B,*d_C;
    int *res = (int*)malloc(sizeT);

    hipMalloc(&d_A, sizeT);       //alokowanie pamięci na kacie graficznej
    hipMalloc(&d_B, sizeT);
    hipMalloc(&d_C, sizeT);

    hipMemcpy(d_A, arrayA.data(), sizeT, hipMemcpyHostToDevice);     // kopiowanie pamięci z RAM do karty
    hipMemcpy(d_B, arrayB.data(), sizeT, hipMemcpyHostToDevice);
    hipMemcpy(d_C, arrayC.data(), sizeT, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(n, n);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
    if (size > 512){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        numBlocks.x = ceil(double(n)/double(threadsPerBlock.x));
        numBlocks.y = ceil(double(n)/double(threadsPerBlock.y));
    }

    multiplication<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n);
    //cudaDeviceSynchronize();
    hipMemcpy(res, d_C, size * sizeof(int), hipMemcpyDeviceToHost);  //na odwrót

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            C[i][j] = res[i * n + j];
        }
    }

    free(res);
}

vector2D generator(int n){
    vector2D t(n,vector1D(n,0));
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            t[i][j]=std::rand()%MAXNumber + MINNumber;
        }
    }
    return t;
}

vector2D multiply(vector2D A,vector2D B){
    int n = A.size();
    int p = A[0].size();
    int m = B[0].size();

    vector2D t(n,vector1D(m,0));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < m; ++j) {
            for (int k = 0; k < p; ++k) {
                t[i][j] += A[i][k] * B[k][j];
            }
        }
    }
    return t;
}

int main(){
    int n=22;
    std::cout<<"GENERATE DATA\n\n";

    vector2D A=generator(n);
    vector2D B=generator(n);
    vector2D C2(n,vector1D(n,0));
    display(A);
    std::cout<<"----------------\n";
    display(B);

    auto start1 = std::chrono::steady_clock::now();
    vector2D C1 = multiply(A,B);
    auto end1 = std::chrono::steady_clock::now();
    std::chrono::duration<double> time1 = end1 - start1;

    std::cout<<"\nRESULT SEQUENCY time: "<<time1.count()*1000<<" ms\n----------------\n";
    display(C1);

    auto start2 = std::chrono::steady_clock::now();
    multiplyCUDA(A,B,C2,n);
    auto end2 = std::chrono::steady_clock::now();
    std::chrono::duration<double> time2 = end2 - start2;

    std::cout<<"\nRESULT CUDA time: "<<time2.count()*1000<<" ms\n----------------\n";
    display(C2);

    A.clear();
    B.clear();
    C1.clear();
    C2.clear();

    return 0;
}
