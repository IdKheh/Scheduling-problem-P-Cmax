
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#define vector2D std::vector<std::vector<int>>
#define vector1D std::vector<int>
#define MAXNumber 10;
#define MINNumber 0;

void display(vector2D t){
    int n = t.size();
    int m = t[0].size();

    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++)
            std::cout<<t[i][j]<<" ";
        std::cout<<std::endl;
    }
}

vector2D generator(int n,int m){
    vector2D t(n,vector1D(m,0));
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            t[i][j]=std::rand()%MAXNumber + MINNumber;
        }
    }
    return t;
}

vector2D multiplay(vector2D A,vector2D B){
    int n = A.size();
    int p = A[0].size();
    int m = B[0].size();

    vector2D t(n,vector1D(m,0));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < m; ++j) {
            for (int k = 0; k < p; ++k) {
                t[i][j] += A[i][k] * B[k][j];
            }
        }
    }
    return t;
}

int main(){
    int n=3, m=1;
    std::cout<<"GENERATE DATA\n\n";

    vector2D A=generator(n,m);
    vector2D B=generator(m,n);
    display(A);
    std::cout<<"----------------\n";
    display(B);

    auto start1 = std::chrono::steady_clock::now();
    vector2D C = multiplay(A,B);
    auto end1 = std::chrono::steady_clock::now();
    std::chrono::duration<double> time1 = end1 - start1;

    std::cout<<"\nRESULT time: "<<time1.count()*1000<<" ms\n----------------\n";
    display(C);

    return 0;
}
