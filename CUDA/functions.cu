#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>
#include <algorithm>
#include "polaczenie.hpp"

void wypisz(std::vector< std::vector<int>> t)
{
    int n=t.size();

    for(int i=0;i<n;i++)
    {
        for(int j=0;j<t[i].size();j++)
            std::cout<<t[i][j]<<" ";
        std::cout<<"\n";
    }
}
void wypisz_male(std::vector<int> t)
{
    int n=t.size();
    for(int i=0;i<n;i++)
        std::cout<<t[i]<<" ";
    
    std::cout<<"\n";
}
int cMax(std::vector<int> machines)
{
    int cmax=0;
    for(int i=0;i<machines.size();i++)
        cmax=std::max(cmax,machines[i]);
    return cmax;
}
